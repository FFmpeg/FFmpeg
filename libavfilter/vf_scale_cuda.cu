
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

extern "C" {

__global__ void Subsample_Bilinear_uchar(hipTextureObject_t uchar_tex,
                                    unsigned char *dst,
                                    int dst_width, int dst_height, int dst_pitch,
                                    int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);
        int y0 = tex2D<unsigned char>(uchar_tex, xi-dx, yi-dy);
        int y1 = tex2D<unsigned char>(uchar_tex, xi+dx, yi-dy);
        int y2 = tex2D<unsigned char>(uchar_tex, xi-dx, yi+dy);
        int y3 = tex2D<unsigned char>(uchar_tex, xi+dx, yi+dy);
        dst[yo*dst_pitch+xo] = (unsigned char)((y0+y1+y2+y3+2) >> 2);
    }
}

__global__ void Subsample_Bilinear_uchar2(hipTextureObject_t uchar2_tex,
                                    uchar2 *dst,
                                    int dst_width, int dst_height, int dst_pitch2,
                                    int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);
        uchar2 c0 = tex2D<uchar2>(uchar2_tex, xi-dx, yi-dy);
        uchar2 c1 = tex2D<uchar2>(uchar2_tex, xi+dx, yi-dy);
        uchar2 c2 = tex2D<uchar2>(uchar2_tex, xi-dx, yi+dy);
        uchar2 c3 = tex2D<uchar2>(uchar2_tex, xi+dx, yi+dy);
        int2 uv;
        uv.x = ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
        uv.y = ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
        dst[yo*dst_pitch2+xo] = make_uchar2((unsigned char)uv.x, (unsigned char)uv.y);
    }
}

__global__ void Subsample_Bilinear_uchar4(hipTextureObject_t uchar4_tex,
                                    uchar4 *dst,
                                    int dst_width, int dst_height, int dst_pitch,
                                    int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);
        uchar4 c0 = tex2D<uchar4>(uchar4_tex, xi-dx, yi-dy);
        uchar4 c1 = tex2D<uchar4>(uchar4_tex, xi+dx, yi-dy);
        uchar4 c2 = tex2D<uchar4>(uchar4_tex, xi-dx, yi+dy);
        uchar4 c3 = tex2D<uchar4>(uchar4_tex, xi+dx, yi+dy);
        int4 res;
        res.x =  ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
        res.y =  ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
        res.z =  ((int)c0.z+(int)c1.z+(int)c2.z+(int)c3.z+2) >> 2;
        res.w =  ((int)c0.w+(int)c1.w+(int)c2.w+(int)c3.w+2) >> 2;
        dst[yo*dst_pitch+xo] = make_uchar4(
            (unsigned char)res.x, (unsigned char)res.y, (unsigned char)res.z, (unsigned char)res.w);
    }
}

__global__ void Subsample_Bilinear_ushort(hipTextureObject_t ushort_tex,
                                    unsigned short *dst,
                                    int dst_width, int dst_height, int dst_pitch,
                                    int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);
        int y0 = tex2D<unsigned short>(ushort_tex, xi-dx, yi-dy);
        int y1 = tex2D<unsigned short>(ushort_tex, xi+dx, yi-dy);
        int y2 = tex2D<unsigned short>(ushort_tex, xi-dx, yi+dy);
        int y3 = tex2D<unsigned short>(ushort_tex, xi+dx, yi+dy);
        dst[yo*dst_pitch+xo] = (unsigned short)((y0+y1+y2+y3+2) >> 2);
    }
}

__global__ void Subsample_Bilinear_ushort2(hipTextureObject_t ushort2_tex,
                                    ushort2 *dst,
                                    int dst_width, int dst_height, int dst_pitch2,
                                    int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);
        ushort2 c0 = tex2D<ushort2>(ushort2_tex, xi-dx, yi-dy);
        ushort2 c1 = tex2D<ushort2>(ushort2_tex, xi+dx, yi-dy);
        ushort2 c2 = tex2D<ushort2>(ushort2_tex, xi-dx, yi+dy);
        ushort2 c3 = tex2D<ushort2>(ushort2_tex, xi+dx, yi+dy);
        int2 uv;
        uv.x = ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
        uv.y = ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
        dst[yo*dst_pitch2+xo] = make_ushort2((unsigned short)uv.x, (unsigned short)uv.y);
    }
}

__global__ void Subsample_Bilinear_ushort4(hipTextureObject_t ushort4_tex,
                                    ushort4 *dst,
                                    int dst_width, int dst_height, int dst_pitch,
                                    int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);
        ushort4 c0 = tex2D<ushort4>(ushort4_tex, xi-dx, yi-dy);
        ushort4 c1 = tex2D<ushort4>(ushort4_tex, xi+dx, yi-dy);
        ushort4 c2 = tex2D<ushort4>(ushort4_tex, xi-dx, yi+dy);
        ushort4 c3 = tex2D<ushort4>(ushort4_tex, xi+dx, yi+dy);
        int4 res;
        res.x =  ((int)c0.x+(int)c1.x+(int)c2.x+(int)c3.x+2) >> 2;
        res.y =  ((int)c0.y+(int)c1.y+(int)c2.y+(int)c3.y+2) >> 2;
        res.z =  ((int)c0.z+(int)c1.z+(int)c2.z+(int)c3.z+2) >> 2;
        res.w =  ((int)c0.w+(int)c1.w+(int)c2.w+(int)c3.w+2) >> 2;
        dst[yo*dst_pitch+xo] = make_ushort4(
            (unsigned short)res.x, (unsigned short)res.y, (unsigned short)res.z, (unsigned short)res.w);
    }
}

}
