
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2018 Philip Langdale <philipl@overt.org>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

template<typename T>
__inline__ __device__ T spatial_predictor(T a, T b, T c, T d, T e, T f, T g,
                                          T h, T i, T j, T k, T l, T m, T n)
{
    int spatial_pred = (d + k)/2;
    int spatial_score = abs(c - j) + abs(d - k) + abs(e - l);

    int score = abs(b - k) + abs(c - l) + abs(d - m);
    if (score < spatial_score) {
        spatial_pred = (c + l)/2;
        spatial_score = score;
        score = abs(a - l) + abs(b - m) + abs(c - n);
        if (score < spatial_score) {
          spatial_pred = (b + m)/2;
          spatial_score = score;
        }
    }
    score = abs(d - i) + abs(e - j) + abs(f - k);
    if (score < spatial_score) {
        spatial_pred = (e + j)/2;
        spatial_score = score;
        score = abs(e - h) + abs(f - i) + abs(g - j);
        if (score < spatial_score) {
          spatial_pred = (f + i)/2;
          spatial_score = score;
        }
    }
    return spatial_pred;
}

__inline__ __device__ int max3(int a, int b, int c)
{
    int x = max(a, b);
    return max(x, c);
}

__inline__ __device__ int min3(int a, int b, int c)
{
    int x = min(a, b);
    return min(x, c);
}

template<typename T>
__inline__ __device__ T temporal_predictor(T A, T B, T C, T D, T E, T F,
                                           T G, T H, T I, T J, T K, T L,
                                           T spatial_pred, bool skip_check)
{
    int p0 = (C + H) / 2;
    int p1 = F;
    int p2 = (D + I) / 2;
    int p3 = G;
    int p4 = (E + J) / 2;

    int tdiff0 = abs(D - I);
    int tdiff1 = (abs(A - F) + abs(B - G)) / 2;
    int tdiff2 = (abs(K - F) + abs(G - L)) / 2;

    int diff = max3(tdiff0, tdiff1, tdiff2);

    if (!skip_check) {
      int maxi = max3(p2 - p3, p2 - p1, min(p0 - p1, p4 - p3));
      int mini = min3(p2 - p3, p2 - p1, max(p0 - p1, p4 - p3));
      diff = max3(diff, mini, -maxi);
    }

    if (spatial_pred > p2 + diff) {
      spatial_pred = p2 + diff;
    }
    if (spatial_pred < p2 - diff) {
      spatial_pred = p2 - diff;
    }

    return spatial_pred;
}

template<typename T>
__inline__ __device__ void yadif_single(T *dst,
                                        hipTextureObject_t prev,
                                        hipTextureObject_t cur,
                                        hipTextureObject_t next,
                                        int dst_width, int dst_height, int dst_pitch,
                                        int src_width, int src_height,
                                        int parity, int tff, bool skip_spatial_check)
{
    // Identify location
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (xo >= dst_width || yo >= dst_height) {
        return;
    }

    // Don't modify the primary field
    if (yo % 2 == parity) {
      dst[yo*dst_pitch+xo] = tex2D<T>(cur, xo, yo);
      return;
    }

    // Calculate spatial prediction
    T a = tex2D<T>(cur, xo - 3, yo - 1);
    T b = tex2D<T>(cur, xo - 2, yo - 1);
    T c = tex2D<T>(cur, xo - 1, yo - 1);
    T d = tex2D<T>(cur, xo - 0, yo - 1);
    T e = tex2D<T>(cur, xo + 1, yo - 1);
    T f = tex2D<T>(cur, xo + 2, yo - 1);
    T g = tex2D<T>(cur, xo + 3, yo - 1);

    T h = tex2D<T>(cur, xo - 3, yo + 1);
    T i = tex2D<T>(cur, xo - 2, yo + 1);
    T j = tex2D<T>(cur, xo - 1, yo + 1);
    T k = tex2D<T>(cur, xo - 0, yo + 1);
    T l = tex2D<T>(cur, xo + 1, yo + 1);
    T m = tex2D<T>(cur, xo + 2, yo + 1);
    T n = tex2D<T>(cur, xo + 3, yo + 1);

    T spatial_pred =
        spatial_predictor(a, b, c, d, e, f, g, h, i, j, k, l, m, n);

    // Calculate temporal prediction
    int is_second_field = !(parity ^ tff);

    hipTextureObject_t prev2 = prev;
    hipTextureObject_t prev1 = is_second_field ? cur : prev;
    hipTextureObject_t next1 = is_second_field ? next : cur;
    hipTextureObject_t next2 = next;

    T A = tex2D<T>(prev2, xo,  yo - 1);
    T B = tex2D<T>(prev2, xo,  yo + 1);
    T C = tex2D<T>(prev1, xo,  yo - 2);
    T D = tex2D<T>(prev1, xo,  yo + 0);
    T E = tex2D<T>(prev1, xo,  yo + 2);
    T F = tex2D<T>(cur,   xo,  yo - 1);
    T G = tex2D<T>(cur,   xo,  yo + 1);
    T H = tex2D<T>(next1, xo,  yo - 2);
    T I = tex2D<T>(next1, xo,  yo + 0);
    T J = tex2D<T>(next1, xo,  yo + 2);
    T K = tex2D<T>(next2, xo,  yo - 1);
    T L = tex2D<T>(next2, xo,  yo + 1);

    spatial_pred = temporal_predictor(A, B, C, D, E, F, G, H, I, J, K, L,
                                      spatial_pred, skip_spatial_check);

    dst[yo*dst_pitch+xo] = spatial_pred;
}

template <typename T>
__inline__ __device__ void yadif_double(T *dst,
                                        hipTextureObject_t prev,
                                        hipTextureObject_t cur,
                                        hipTextureObject_t next,
                                        int dst_width, int dst_height, int dst_pitch,
                                        int src_width, int src_height,
                                        int parity, int tff, bool skip_spatial_check)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (xo >= dst_width || yo >= dst_height) {
        return;
    }

    if (yo % 2 == parity) {
      // Don't modify the primary field
      dst[yo*dst_pitch+xo] = tex2D<T>(cur, xo, yo);
      return;
    }

    T a = tex2D<T>(cur, xo - 3, yo - 1);
    T b = tex2D<T>(cur, xo - 2, yo - 1);
    T c = tex2D<T>(cur, xo - 1, yo - 1);
    T d = tex2D<T>(cur, xo - 0, yo - 1);
    T e = tex2D<T>(cur, xo + 1, yo - 1);
    T f = tex2D<T>(cur, xo + 2, yo - 1);
    T g = tex2D<T>(cur, xo + 3, yo - 1);

    T h = tex2D<T>(cur, xo - 3, yo + 1);
    T i = tex2D<T>(cur, xo - 2, yo + 1);
    T j = tex2D<T>(cur, xo - 1, yo + 1);
    T k = tex2D<T>(cur, xo - 0, yo + 1);
    T l = tex2D<T>(cur, xo + 1, yo + 1);
    T m = tex2D<T>(cur, xo + 2, yo + 1);
    T n = tex2D<T>(cur, xo + 3, yo + 1);

    T spatial_pred;
    spatial_pred.x =
        spatial_predictor(a.x, b.x, c.x, d.x, e.x, f.x, g.x, h.x, i.x, j.x, k.x, l.x, m.x, n.x);
    spatial_pred.y =
        spatial_predictor(a.y, b.y, c.y, d.y, e.y, f.y, g.y, h.y, i.y, j.y, k.y, l.y, m.y, n.y);

    // Calculate temporal prediction
    int is_second_field = !(parity ^ tff);

    hipTextureObject_t prev2 = prev;
    hipTextureObject_t prev1 = is_second_field ? cur : prev;
    hipTextureObject_t next1 = is_second_field ? next : cur;
    hipTextureObject_t next2 = next;

    T A = tex2D<T>(prev2, xo,  yo - 1);
    T B = tex2D<T>(prev2, xo,  yo + 1);
    T C = tex2D<T>(prev1, xo,  yo - 2);
    T D = tex2D<T>(prev1, xo,  yo + 0);
    T E = tex2D<T>(prev1, xo,  yo + 2);
    T F = tex2D<T>(cur,   xo,  yo - 1);
    T G = tex2D<T>(cur,   xo,  yo + 1);
    T H = tex2D<T>(next1, xo,  yo - 2);
    T I = tex2D<T>(next1, xo,  yo + 0);
    T J = tex2D<T>(next1, xo,  yo + 2);
    T K = tex2D<T>(next2, xo,  yo - 1);
    T L = tex2D<T>(next2, xo,  yo + 1);

    spatial_pred.x =
        temporal_predictor(A.x, B.x, C.x, D.x, E.x, F.x, G.x, H.x, I.x, J.x, K.x, L.x,
                           spatial_pred.x, skip_spatial_check);
    spatial_pred.y =
        temporal_predictor(A.y, B.y, C.y, D.y, E.y, F.y, G.y, H.y, I.y, J.y, K.y, L.y,
                           spatial_pred.y, skip_spatial_check);

    dst[yo*dst_pitch+xo] = spatial_pred;
}

extern "C" {

__global__ void yadif_uchar(unsigned char *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check)
{
    yadif_single(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check);
}

__global__ void yadif_ushort(unsigned short *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check)
{
    yadif_single(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check);
}

__global__ void yadif_uchar2(uchar2 *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check)
{
    yadif_double(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check);
}

__global__ void yadif_ushort2(ushort2 *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check)
{
    yadif_double(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check);
}

} /* extern "C" */
