
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

extern "C" {

__global__ void Thumbnail_uchar(hipTextureObject_t uchar_tex,
                                int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < src_height && x < src_width)
    {
        unsigned char pixel = tex2D<unsigned char>(uchar_tex, x, y);
        atomicAdd(&histogram[pixel], 1);
    }
}

__global__ void Thumbnail_uchar2(hipTextureObject_t uchar2_tex,
                                 int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        uchar2 pixel = tex2D<uchar2>(uchar2_tex, x, y);
        atomicAdd(&histogram[pixel.x], 1);
        atomicAdd(&histogram[256 + pixel.y], 1);
    }
}

__global__ void Thumbnail_ushort(hipTextureObject_t ushort_tex,
                                 int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        unsigned short pixel = (tex2D<unsigned short>(ushort_tex, x, y) + 128) >> 8;
        atomicAdd(&histogram[pixel], 1);
    }
}

__global__ void Thumbnail_ushort2(hipTextureObject_t ushort2_tex,
                                  int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        ushort2 pixel = tex2D<ushort2>(ushort2_tex, x, y);
        atomicAdd(&histogram[(pixel.x + 128) >> 8], 1);
        atomicAdd(&histogram[256 + (pixel.y + 128) >> 8], 1);
    }
}

}
