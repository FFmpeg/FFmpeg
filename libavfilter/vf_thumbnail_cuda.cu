#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;

__global__ void Thumbnail_uchar(int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < src_height && x < src_width)
    {
        unsigned char pixel = tex2D(uchar_tex, x, y);
        atomicAdd(&histogram[pixel], 1);
    }
}

__global__ void Thumbnail_uchar2(int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        uchar2 pixel = tex2D(uchar2_tex, x, y);
        atomicAdd(&histogram[pixel.x], 1);
        atomicAdd(&histogram[256 + pixel.y], 1);
    }
}

__global__ void Thumbnail_ushort(int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        unsigned short pixel = (tex2D(ushort_tex, x, y) + 128) >> 8;
        atomicAdd(&histogram[pixel], 1);
    }
}

__global__ void Thumbnail_ushort2(int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        ushort2 pixel = tex2D(ushort2_tex, x, y);
        atomicAdd(&histogram[(pixel.x + 128) >> 8], 1);
        atomicAdd(&histogram[256 + (pixel.y + 128) >> 8], 1);
    }
}

}
