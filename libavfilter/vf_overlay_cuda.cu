
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2020 Yaroslav Pogrebnyak <yyyaroslav@gmail.com>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

extern "C" {

__global__ void Overlay_Cuda(
    int x_position, int y_position,
    unsigned char* main, int main_linesize,
    unsigned char* overlay, int overlay_linesize,
    int overlay_w, int overlay_h,
    unsigned char* overlay_alpha, int alpha_linesize,
    int alpha_adj_x, int alpha_adj_y)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= overlay_w + x_position ||
        y >= overlay_h + y_position ||
        x < x_position ||
        y < y_position ) {

        return;
    }

    int overlay_x = x - x_position;
    int overlay_y = y - y_position;

    float alpha = 1.0;
    if (alpha_linesize) {
        alpha = overlay_alpha[alpha_adj_x * overlay_x  + alpha_adj_y * overlay_y * alpha_linesize] / 255.0f;
    }

    main[x + y*main_linesize] = alpha * overlay[overlay_x + overlay_y * overlay_linesize] + (1.0f - alpha) * main[x + y*main_linesize];
}

}

